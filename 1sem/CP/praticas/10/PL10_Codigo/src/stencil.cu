#include "hip/hip_runtime.h"
#include "stencil.h"

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

using namespace std;

__global__ 
void stencilKernel (float *a, float *c) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// initialise the array with the results
	c[id] = 0;

	// iterate through the neighbours required to calculate
	// the values for the current position of c
	//for (int n = ?; n <= ?; n++) {
	//	??
	//}

}

void stencil (float *a, float *c) {
	chrono::steady_clock::time_point begin = chrono::steady_clock::now();

	for (int i = 0; i < SIZE; i++) {
		// considers 4 neighbours
		for (int n = -2; n <= 2; n++) {
			if ((i + n >= 0) && (i + n < SIZE))
				c[i] += a[i + n];
		}
	}

	chrono::steady_clock::time_point end = chrono::steady_clock::now();
	cout << endl << "Sequential CPU execution: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl << endl;
}

void launchStencilKernel (float *a, float *c) {
	// pointers to the device memory
	float *da, *dc;
	// declare variable with size of the array in bytes
	int bytes = SIZE * sizeof(float);

	// allocate the memory on the device
	hipMalloc ((void**) &da, bytes);
	hipMalloc (??);
	checkCUDAError("mem allocation");

	// copy inputs to the device
	hipMemcpy (??);
	checkCUDAError("memcpy h->d");

	// launch the kernel
	startKernelTime ();
	stencilKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (??, ??);
	stopKernelTime ();
	checkCUDAError("kernel invocation");

	// copy the output to the host
	hipMemcpy (??);
	checkCUDAError("memcpy d->h");

	// free the device memory
	hipFree(da);
	hipFree(dc);
	checkCUDAError("mem free");
}

int main( int argc, char** argv) {
	// arrays on the host
	float a[SIZE], b[SIZE], c[SIZE];

	// initialises the array
	for (unsigned i = 0; i < SIZE; ++i)
		a[i] = (float) rand() / RAND_MAX;

	stencil (a, b);
	
	launchStencilKernel (a, c);

	return 0;
}
